#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(int* output, size_t pitch, int width, int height, float lowerX, float lowerY, float stepX, float stepY, 
    int maxIterations, int group_size) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

  int i = (blockIdx.x * blockDim.x + threadIdx.x) * group_size;
  int j = (blockIdx.y * blockDim.y + threadIdx.y) * group_size;

  for(int gi=0; gi < group_size; gi++) {
    for(int gj=0; gj < group_size; gj++) {
        float x = lowerX + (gi + i) * stepX;
        float y = lowerY + (gj + j) * stepY;

        int *ele = (int*)((char*)output + (gj + j) * pitch) + ( gi + i  );
        *ele = mandel(x, y, maxIterations);	
    }
  }
}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;
	
  int* cuda_output;
  int* output;
   hipHostAlloc((void**)&output, resX * resY * sizeof(int) , hipHostMallocDefault) ;
  size_t pitch;
  hipMallocPitch((void**)&cuda_output, &pitch, resX * sizeof(int), resY);

  int block_size = 8;
  int group_size = 2; // 1 thread process 4 pixel
  dim3 blocksize(block_size, block_size);
  dim3 gridsize(resX / block_size / group_size, resY / block_size / group_size);

  mandelKernel<<<gridsize, blocksize>>>(cuda_output, pitch, resX, resY, lowerX, lowerY, stepX, stepY, maxIterations, group_size );
  hipMemcpy2D(output, resX * sizeof(int), cuda_output, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

  memcpy(img, output, resX*resY*sizeof(int));

  hipFree(cuda_output);
  hipHostFree(output);
}
