#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__device__ int mandel(float c_re, float c_im, int count)
{
  float z_re = c_re, z_im = c_im;
  int i;
  for (i = 0; i < count; ++i)
  {

    if (z_re * z_re + z_im * z_im > 4.f)
      break;

    float new_re = z_re * z_re - z_im * z_im;
    float new_im = 2.f * z_re * z_im;
    z_re = c_re + new_re;
    z_im = c_im + new_im;
  }

  return i;
}

__global__ void mandelKernel(int* output, size_t pitch, int width, int height, float lowerX, float lowerY, float stepX, float stepY, int maxIterations) {
    // To avoid error caused by the floating number, use the following pseudo code
    //
    // float x = lowerX + thisX * stepX;
    // float y = lowerY + thisY * stepY;

	int i = blockIdx.x * blockDim.x + threadIdx.x;
  int j = blockIdx.y * blockDim.y + threadIdx.y;
  if (i >= width || j >= height ) {
    return;
  }
    float x = lowerX + i * stepX;
    float y = lowerY + j * stepY;
    int index = j * pitch + i;
    int *ele = (int*)((char*)output + j * pitch) + i;
    *ele = mandel(x, y, maxIterations);	

}

// Host front-end function that allocates the memory and launches the GPU kernel
void hostFE (float upperX, float upperY, float lowerX, float lowerY, int* img, int resX, int resY, int maxIterations)
{
  float stepX = (upperX - lowerX) / resX;
  float stepY = (upperY - lowerY) / resY;
	
  int* cuda_output;
  int* output;
   hipHostAlloc((void**)&output, resX * resY * sizeof(int) , hipHostMallocDefault) ;
  size_t pitch;
  hipMallocPitch((void**)&cuda_output, &pitch, resX * sizeof(int), resY);
  int block_size_x = 8;
  int block_size_y = 8;

	dim3 blocksize(block_size_x, block_size_y);
	dim3 gridsize(resX / block_size_x , resY / block_size_y );

  mandelKernel<<<gridsize, blocksize>>>(cuda_output, pitch, resX, resY, lowerX, lowerY, stepX, stepY, maxIterations );
  hipMemcpy2D(output, resX * sizeof(int), cuda_output, pitch, resX * sizeof(int), resY, hipMemcpyDeviceToHost);

  memcpy(img, output, resX*resY*sizeof(int));

	hipFree(cuda_output);
	hipHostFree(output);
}


